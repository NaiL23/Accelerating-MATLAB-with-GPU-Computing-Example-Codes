#include "mex.h"
#include <hipblas.h>
// To compile: mexcuda cublasDemo.cu -lcublas

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray *prhs[])
{
    if (nrhs != 2)
        mexErrMsgTxt("Invaid number of input arguments");
   
    if (!mxIsSingle(prhs[0]) || !mxIsSingle(prhs[1]))
        mexErrMsgTxt("input matrices must be single");
    
    float* A = (float*)mxGetData(prhs[0]);
    float* B = (float*)mxGetData(prhs[1]);
    
    int numARows = mxGetM(prhs[0]);
    int numACols = mxGetN(prhs[0]);
    int numBRows = mxGetM(prhs[1]);
    int numBCols = mxGetN(prhs[1]);
    int numCRows = numARows;
    int numCCols = numBCols;
    
    plhs[0] = mxCreateNumericMatrix(numCRows, numCCols, mxSINGLE_CLASS, mxREAL);
    float* C = (float*)mxGetData(plhs[0]);
    
    float *deviceA, *deviceB, *deviceC;
    hipMalloc(&deviceA, sizeof(float) * numARows * numACols);
    hipMalloc(&deviceB, sizeof(float) * numBRows * numBCols);
    hipMalloc(&deviceC, sizeof(float) * numCRows * numCCols);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMatrix(numARows,
                    numACols,
                    sizeof(float),
                    A,
                    numARows,
                    deviceA,
                    numARows);
    hipblasSetMatrix(numBRows,
                    numBCols,
                    sizeof(float),
                    B,
                    numBRows,
                    deviceB,
                    numBRows);

    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                numARows, numBCols, numACols,
                &alpha, 
                deviceA, numARows,
                deviceB, numBRows,
                &beta,
                deviceC, numCRows);
    
    hipblasGetMatrix(numCRows,
                    numCCols,
                    sizeof(float),
                    deviceC,
                    numCRows,
                    C,
                    numCRows);
    
    hipblasDestroy(handle);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}