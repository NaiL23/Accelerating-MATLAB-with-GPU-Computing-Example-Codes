#include "mex.h"
#include <hipfft/hipfft.h>

// To compile: mexcuda cufftDemo.cu -lcufft

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray *prhs[])
{
    if (nrhs != 1)
        mexErrMsgTxt("Invaid number of input arguments");

    if (!mxIsSingle(prhs[0]) && !mxIsSingle(prhs[1]))
        mexErrMsgTxt("input data type must be single");

    float* A = (float*)mxGetData(prhs[0]);

    int numARows = mxGetM(prhs[0]);
    int numACols = mxGetN(prhs[0]);

    float *deviceA;

    hipMalloc(&deviceA, sizeof(float) * numARows * numACols);
    hipMemcpy(deviceA, A, numARows * numACols * sizeof(float),
               hipMemcpyHostToDevice);
 
    int outRows = numARows /2 + 1;
    int outCols = numACols;
    hipfftComplex* deviceOut;
    hipMalloc(&deviceOut, sizeof(hipfftComplex) * outRows * outCols);

    hipfftHandle plan;
    hipfftPlan2d(&plan, numACols, numARows, HIPFFT_R2C);
    hipfftExecR2C(plan, deviceA, deviceOut);

    float* out = (float*)mxMalloc(sizeof(hipfftComplex) * outRows * outCols);
    hipMemcpy(out, deviceOut, outRows * outCols * sizeof(hipfftComplex),
               hipMemcpyDeviceToHost);

    plhs[0] = mxCreateNumericMatrix(outRows, outCols, mxSINGLE_CLASS, mxCOMPLEX);
    float* real = (float*)mxGetPr(plhs[0]);
    float* imag = (float*)mxGetPi(plhs[0]);
    float* complex = out;
    for (int c = 0; c < outCols; ++c)
    {
        for (int r = 0; r < outRows; ++r)
        {
            *real++ = *complex++;
            *imag++ = *complex++;
        }
    }

    mxFree(out);
    hipfftDestroy(plan);
    hipFree(deviceA);
    hipDeviceReset();
}
